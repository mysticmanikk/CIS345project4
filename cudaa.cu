#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define EPSILON (0.001f)
#define notEqual(x,y)     (fabs((x) - (y)) > EPSILON)

typedef unsigned long long SysClock;

static SysClock currentTicks() {
    timespec spec;
    clock_gettime(CLOCK_THREAD_CPUTIME_ID, &spec);
    return (SysClock)((float)(spec.tv_sec) * 1e9 + (float)(spec.tv_nsec));
}

static double secondsPerTick() {
    static int initialized = 0;
    static double secondsPerTick_val;
    if (initialized) return secondsPerTick_val;
    FILE *fp = fopen("/proc/cpuinfo","r");
    char input[1024];
    if (!fp) {
        fprintf(stderr, "resetScale failed: couldn't find /proc/cpuinfo.");
	exit(-1);
    }
    secondsPerTick_val = 1e-9;
    while (!feof(fp) && fgets(input, 1024, fp)) {
        float GHz, MHz;
	if (strstr(input, "model name")) {
	    char* at_sign = strstr(input, "@");
	    if (at_sign) {
	        char* after_at = at_sign + 1;
		char* GHz_str = strstr(after_at, "GHz");
		char* MHz_str = strstr(after_at, "MHz");
		if (GHz_str) {
		    *GHz_str = '\0';
		    if (1 == sscanf(after_at, "%f", &GHz)) {
		        //printf("GHz = %f\n", GHz);
			secondsPerTick_val = 1e-9f / GHz;
			break;
		    }
		} else if (MHz_str) {
		    *MHz_str = '\0';
		    if (1 == sscanf(after_at, "%f", &MHz)) {
		        //printf("MHz = %f\n", MHz);
			secondsPerTick_val = 1e-6f / GHz;
			break;
		    }
		}
	    }
	} else if (1 == sscanf(input, "cpu MHz : %f", &MHz)) {
	    //printf("MHz = %f\n", MHz);
	    secondsPerTick_val = 1e-6f / MHz;
	    break;
	}
    }
    fclose(fp);
    initialized = 1;
    return secondsPerTick_val;
}

static double currentSeconds() {
    return currentTicks() * secondsPerTick();
}

float toBW(int bytes, float sec) {
  return (float)(bytes) / (1024. * 1024. * 1024.) / sec;
}

__global__ void faxpy_1blk_kernel(int N, float alpha, float *x, float *y, float *result) {
    // TODO insert your CUDA kernel code here
    // TODO one block of threads
}

__global__ void faxpy_mblk_kernel(int N, float alpha, float* x, float* y, float* result) {

    // TODO insert your CUDA kernel code here
    // TODO multi-blocks of threads
}

void faxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    //
    // TODO allocate device memory buffers on the GPU using hipMalloc
    //


    // start timing after allocation of device memory
    double startTime = currentSeconds();

    //
    // TODO copy input arrays to the GPU using hipMemcpy
    //

    double midTime1 = currentSeconds();

    //
    // TODO run kernel, either 1-block kernel or multi-block kernel
    //

    // IMPORTANT, wait for the completion at GPU
    hipDeviceSynchronize();

    double midTime2 = currentSeconds();

    //
    // TODO copy result from GPU using hipMemcpy
    //

    // end timing after result has been copied back into host memory
    double endTime = currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    double transferDur = midTime1 - startTime;
    printf("xy array --> device %.3f ms\n", 1000.f * transferDur);

    double gpu_compute_dur = midTime2 - midTime1;
    printf("GPU computation duration %.3f ms\n", 1000.f * gpu_compute_dur);

    // TODO free memory buffers on the GPU

}

void faxpyCPU(int N, float alpha, float *xarray, float *yarray, float *resultarray) {
    double startTime = currentSeconds();
    for (int i = 0; i < N; i++) {
        resultarray[i] = alpha * xarray[i] + yarray[i];
    }
    double endTime = currentSeconds();
    double cpu_dur = endTime - startTime;
    printf("CPU computation duration %.3f ms\n", 1000.f * cpu_dur);
}

int main(int argc, char** argv)
{

    int N = 20 * 1000 * 1000;

    const float alpha = 5.0f;
    const float max = 999.0f;
    float* xarray = (float *)malloc(sizeof(float)*N);
    float* yarray = (float *)malloc(sizeof(float)*N);
    float* resultarray = (float *)malloc(sizeof(float)*N);
    float* checkarray = (float *)malloc(sizeof(float)*N);

    for (int i=0; i<N; i++) {
	xarray[i] = ((float)rand()/(float)(RAND_MAX)) * max;
	yarray[i] = ((float)rand()/(float)(RAND_MAX)) * max;
        resultarray[i] = 0.f;
    }

    faxpyCuda(N, alpha, xarray, yarray, resultarray);

    faxpyCPU(N, alpha, xarray, yarray, checkarray);

    // Verify the FAXPY computatin at GPU is correct
    for (int i = 0; i < N; i++) {
      if (notEqual(checkarray[i], resultarray[i])) {
        fprintf(stderr, "Error: device axpy outputs incorrect result."
			" A[%d] = %.5f, expecting %.5f.\n", i, resultarray[i], checkarray[i]);
	exit(1);
      }
    }
    printf("device faxpy outputs are correct!\n");

    free(xarray);
    free(yarray);
    free(resultarray);
    free(checkarray);

    return 0;
}

